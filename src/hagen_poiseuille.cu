#include "hip/hip_runtime.h"
#include <cassert>
#include <glm/gtc/type_ptr.hpp>
#include <iostream>
#include <limits>
#include <random>

#include "alluvion/colormaps.hpp"
#include "alluvion/constants.hpp"
#include "alluvion/dg/infinite_cylinder_distance.hpp"
#include "alluvion/dg/sphere_distance.hpp"
#include "alluvion/float_shorthands.hpp"
#include "alluvion/pile.hpp"
#include "alluvion/runner.hpp"
#include "alluvion/solver_df.hpp"
#include "alluvion/store.hpp"
#include "alluvion/typesetter.hpp"

using namespace alluvion;
using namespace alluvion::dg;

int main(void) {
  Store store;
  Display* display = store.create_display(800, 600, "particle view");
  Runner runner;

  F particle_radius = 0.0025_F;
  F kernel_radius = particle_radius * 4.0_F;
  F density0 = 1000.0_F;
  F cubical_particle_volume =
      8 * particle_radius * particle_radius * particle_radius;
  F volume_relative_to_cube = 0.8_F;
  F particle_mass =
      cubical_particle_volume * volume_relative_to_cube * density0;

  store.get_cn<F>().set_cubic_discretization_constants();
  store.get_cn<F>().set_kernel_radius(kernel_radius);
  store.get_cn<F>().set_particle_attr(particle_radius, particle_mass, density0);
  store.get_cn<F>().axial_gravity = -10.0_F;
  store.get_cn<F>().radial_gravity = 5.0_F;
  store.get_cn<F>().set_boundary_epsilon(1e-9_F);
  F viscosity = 5e-6_F;
  F vorticity = 0.01_F;
  F inertia_inverse = 0.1_F;
  F viscosity_omega = 0.5_F;
  F surface_tension_coeff = 0.05_F;
  F surface_tension_boundary_coeff = 0.01_F;
  store.get_cn<F>().viscosity = viscosity;

  I kM = 5;
  F cylinder_length = 2._F * kM * kernel_radius;
  I kQ = 5;
  F R = kernel_radius * kQ;

  const char* font_filename = "/usr/share/fonts/truetype/freefont/FreeMono.ttf";
  Typesetter typesetter(display, font_filename, 0, 30);
  typesetter.load_ascii();

  display->camera_.setEye(0._F, 0._F, R * 6._F);
  display->camera_.setClipPlanes(particle_radius * 10._F, R * 20._F);
  display->update_trackball_camera();

  GLuint colormap_tex =
      display->create_colormap(kViridisData.data(), kViridisData.size());

  GLuint glyph_quad = display->create_dynamic_array_buffer<float4>(6, nullptr);

  // rigids
  F restitution = 1._F;
  F friction = 0._F;
  F boundary_viscosity = viscosity * 1.5_F;
  U max_num_contacts = 512;
  Pile<F3, Q, F> pile(store, max_num_contacts);
  pile.add(new InfiniteCylinderDistance<F3, F>(R), U3{64, 1, 64}, -1._F, 0,
           Mesh(), 0._F, restitution, friction, boundary_viscosity, F3{1, 1, 1},
           F3{0, 0, 0}, Q{0, 0, 0, 1}, Mesh());
  pile.build_grids(4 * kernel_radius);
  pile.reallocate_kinematics_on_device();
  store.get_cn<F>().contact_tolerance = particle_radius;

  // particles
  U naive_num_particles =
      static_cast<U>(kPi<F> * (R - particle_radius) * (R - particle_radius) *
                     cylinder_length * density0 / particle_mass);
  U max_num_particles = static_cast<U>(kPi<F> * R * R * cylinder_length *
                                       density0 / particle_mass);
  // grid
  U3 grid_res{static_cast<U>(kQ * 2), static_cast<U>(kM * 2),
              static_cast<U>(kQ * 2)};
  I3 grid_offset{-kQ, -kM, -kQ};
  U max_num_particles_per_cell = 64;
  U max_num_neighbors_per_particle = 64;
  store.get_cni().grid_res = grid_res;
  store.get_cni().grid_offset = grid_offset;
  store.get_cni().max_num_particles_per_cell = max_num_particles_per_cell;
  store.get_cni().max_num_neighbors_per_particle =
      max_num_neighbors_per_particle;
  store.get_cn<F>().set_wrap_length(grid_res.y * kernel_radius);

  std::unique_ptr<GraphicalVariable<1, F3>> particle_x(
      store.create_graphical<1, F3>({max_num_particles}));
  std::unique_ptr<GraphicalVariable<1, F>> particle_normalized_attr(
      store.create_graphical<1, F>({max_num_particles}));
  Variable<1, F3> particle_v = store.create<1, F3>({max_num_particles});
  Variable<1, F3> particle_a = store.create<1, F3>({max_num_particles});
  Variable<1, F> particle_density = store.create<1, F>({max_num_particles});
  Variable<2, F3> particle_boundary_xj =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<2, F> particle_boundary_volume =
      store.create<2, F>({pile.get_size(), max_num_particles});
  Variable<2, F3> particle_force =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<2, F3> particle_torque =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<1, F> particle_cfl_v2 = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_dfsph_factor =
      store.create<1, F>({max_num_particles});
  Variable<1, F> particle_kappa = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_kappa_v = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_density_adv = store.create<1, F>({max_num_particles});
  Variable<4, Q> pid = store.create<4, Q>(
      {grid_res.x, grid_res.y, grid_res.z, max_num_particles_per_cell});
  Variable<3, U> pid_length =
      store.create<3, U>({grid_res.x, grid_res.y, grid_res.z});
  Variable<2, Q> particle_neighbors =
      store.create<2, Q>({max_num_particles, max_num_neighbors_per_particle});
  Variable<1, U> particle_num_neighbors =
      store.create<1, U>({max_num_particles});

  SolverDf<F3, Q, F> solver_df(
      runner, pile, *particle_x, *particle_normalized_attr, particle_v,
      particle_a, particle_density, particle_boundary_xj,
      particle_boundary_volume, particle_force, particle_torque,
      particle_cfl_v2, particle_dfsph_factor, particle_kappa, particle_kappa_v,
      particle_density_adv, pid, pid_length, particle_neighbors,
      particle_num_neighbors);
  solver_df.dt = 1e-3_F;
  solver_df.max_dt = 1e-3_F;
  solver_df.min_dt = 0.0_F;
  solver_df.cfl = 2e-2_F;
  solver_df.particle_radius = particle_radius;

  U initial_num_particles = naive_num_particles;
  F slice_distance = particle_radius * 2._F;
  U num_slices = static_cast<U>(cylinder_length / slice_distance);
  U num_particles_per_slice = initial_num_particles / num_slices;
  initial_num_particles = num_particles_per_slice * num_slices;

  store.copy_cn<F>();
  store.map_graphical_pointers();
  Runner::launch(initial_num_particles, 256, [&](U grid_size, U block_size) {
    create_fluid_cylinder<F3, F><<<grid_size, block_size>>>(
        *particle_x, initial_num_particles, R - particle_radius * 2._F,
        num_particles_per_slice, particle_radius * 2._F,
        cylinder_length * -0.5_F);
  });
  store.unmap_graphical_pointers();
  solver_df.num_particles = initial_num_particles;

  U step_id = 0;
  F t = 0;
  F max_density_error = std::numeric_limits<F>::max();
  F min_density_error = std::numeric_limits<F>::max();
  F naive_filled_percentage = 0;
  bool should_close = false;

  F max_particle_speed = 99.9_F;
  F min_particle_speed = 99.9_F;
  F sum_particle_velocity_components = 99.9_F;
  F last_stationary_t = 0._F;

  F next_emission_t = 5.0_F;
  F last_emission_t = 0._F;
  bool speed_ready_before_emission = false;

  std::random_device rd{};
  std::mt19937 gen{rd()};
  std::normal_distribution<F> d{0, 0.05};

  U2 target_num_particles_range = U2{0, 7213};
  U last_saved_num_particles = 0;

  // density sample points
  U num_sample_slices = 32;
  U num_samples_per_slice = 16;
  U num_samples = num_sample_slices * num_samples_per_slice;
  Variable<1, F3> sample_x = store.create<1, F3>({num_samples});
  Variable<1, F> sample_data1 = store.create<1, F>({num_samples});
  std::vector<F> sample_data1_host(num_samples);
  Variable<2, Q> sample_neighbors =
      store.create<2, Q>({num_samples, max_num_neighbors_per_particle});
  Variable<1, U> sample_num_neighbors = store.create<1, U>({num_samples});
  Variable<2, F3> sample_boundary_xj =
      store.create<2, F3>({pile.get_size(), num_samples});
  Runner::launch(initial_num_particles, 256, [&](U grid_size, U block_size) {
    create_fluid_cylinder<F3, F><<<grid_size, block_size>>>(
        sample_x, num_samples, R - particle_radius * 2._F,
        num_samples_per_slice, cylinder_length / num_sample_slices,
        cylinder_length * -0.5_F);
  });
  std::vector<F3> sample_x_host(num_samples);
  sample_x.get_bytes(sample_x_host.data());
  assert(sample_x_host[0].x == sample_x_host[2].z == 0);
  assert(sample_x_host[0].y == -cylinder_length / 2);

  display->add_shading_program(new ShadingProgram(
      nullptr, nullptr, {}, {}, [&](ShadingProgram& program, Display& display) {
        if (should_close) {
          return;
        }

        store.map_graphical_pointers();
        for (U frame_interstep = 0; frame_interstep < 10; ++frame_interstep) {
          if (t > 10._F && solver_df.num_particles == 2704 &&
              t - last_stationary_t > 0.2_F) {
            // particle_x->write_file("x5-2704.alu", solver_df.num_particles);
            // should_close = true;
          }
          if (min_particle_speed > 2._F || (step_id % 10000 == 0)) {
            particle_v.set_zero();
            particle_dfsph_factor.set_zero();
            particle_kappa.set_zero();
            particle_kappa_v.set_zero();
            particle_density_adv.set_zero();
            last_stationary_t = t;
            std::cout << "last stationary t = " << last_stationary_t
                      << std::endl;
          }
          if (!speed_ready_before_emission) {
            if (min_density_error < -0.5) {
              if (max_particle_speed < 1e-1_F)
                speed_ready_before_emission = true;
            } else if (min_density_error < -0.05) {
              if (max_particle_speed < 5e-3_F)
                speed_ready_before_emission = true;
            } else {
              if (max_particle_speed < 1e-2_F && min_density_error > -1e-2 &&
                  max_density_error < 1e-2) {
                speed_ready_before_emission = true;
              }
            }
          }
          if (solver_df.num_particles >= target_num_particles_range.x &&
              solver_df.num_particles <= target_num_particles_range.y &&
              solver_df.num_particles != last_saved_num_particles) {
            speed_ready_before_emission = false;
            if (max_particle_speed < 2e-3 && (t - last_stationary_t > 4)) {
              std::stringstream filename_stream;
              filename_stream << "x" << kQ << "-" << kM << "-"
                              << solver_df.num_particles << ".alu";
              std::string filename = filename_stream.str();
              particle_x->write_file(filename.c_str(), solver_df.num_particles);
              last_saved_num_particles = solver_df.num_particles;
            }
          }
          if (t > next_emission_t && speed_ready_before_emission &&
              solver_df.num_particles < max_num_particles) {
            F3 new_particle_pos{0, cylinder_length / 2 - 1e-4, 0};
            F3 new_particle_v{d(gen), d(gen), d(gen)};
            next_emission_t = t + particle_radius * 2 / length(new_particle_v);
            particle_x->set_bytes(&new_particle_pos, sizeof(F3),
                                  sizeof(F3) * solver_df.num_particles);
            particle_v.set_bytes(&new_particle_v, sizeof(F3),
                                 sizeof(F3) * solver_df.num_particles);
            ++solver_df.num_particles;
            last_emission_t = t;
            speed_ready_before_emission = false;
          }

          solver_df.step<1, 1>();

          t += solver_df.dt;
          step_id += 1;

          max_density_error =
              Runner::max(particle_density, solver_df.num_particles) /
                  density0 -
              1;
          min_density_error =
              Runner::min(particle_density, solver_df.num_particles) /
                  density0 -
              1;
          max_particle_speed =
              sqrt(Runner::max(particle_cfl_v2, solver_df.num_particles));
          min_particle_speed =
              sqrt(Runner::min(particle_cfl_v2, solver_df.num_particles));
          sum_particle_velocity_components =
              Runner::sum<F>(particle_v.ptr_, particle_v.get_num_primitives());

          F expected_total_volume = kPi<F> * (R - particle_radius) *
                                    (R - particle_radius) * cylinder_length;
          naive_filled_percentage = solver_df.num_particles * particle_mass /
                                    density0 / expected_total_volume;
          if (step_id % 100 == 0) {
            pid_length.set_zero();
            Runner::launch(
                solver_df.num_particles, 256, [&](U grid_size, U block_size) {
                  update_particle_grid<<<grid_size, block_size>>>(
                      *particle_x, pid, pid_length, solver_df.num_particles);
                });
            Runner::launch(num_samples, 256, [&](U grid_size, U block_size) {
              make_neighbor_list<1><<<grid_size, block_size>>>(
                  sample_x, pid, pid_length, sample_neighbors,
                  sample_num_neighbors, num_samples);
            });
            Runner::launch(
                solver_df.num_particles, 256, [&](U grid_size, U block_size) {
                  compute_density<<<grid_size, block_size>>>(
                      *particle_x, particle_neighbors, particle_num_neighbors,
                      particle_density, particle_boundary_xj,
                      particle_boundary_volume, solver_df.num_particles);
                });
            Runner::launch(num_samples, 256, [&](U grid_size, U block_size) {
              sample_fluid<<<grid_size, block_size>>>(
                  sample_x, *particle_x, particle_density, particle_density,
                  sample_neighbors, sample_num_neighbors, sample_data1,
                  num_samples);
            });
            sample_data1.get_bytes(sample_data1_host.data());
            for (U i = 0; i < num_samples; i += 127) {
              std::cout << sample_data1_host[i] << " ";
            }
            std::cout << std::endl;
          }
        }
        solver_df.colorize_speed(0, 2.0);
        store.unmap_graphical_pointers();
      }));

  // {{{
#include "alluvion/glsl/particle.frag"
#include "alluvion/glsl/particle.vert"
  display->add_shading_program(new ShadingProgram(
      kParticleVertexShaderStr, kParticleFragmentShaderStr,
      {"particle_radius", "screen_dimension", "M", "V", "P",
       "camera_worldspace", "material.specular", "material.shininess",
       "directional_light.direction", "directional_light.ambient",
       "directional_light.diffuse", "directional_light.specular",
       "point_lights[0].position", "point_lights[0].constant",
       "point_lights[0].linear", "point_lights[0].quadratic",
       "point_lights[0].ambient", "point_lights[0].diffuse",
       "point_lights[0].specular",
       //
       "point_lights[1].position", "point_lights[1].constant",
       "point_lights[1].linear", "point_lights[1].quadratic",
       "point_lights[1].ambient", "point_lights[1].diffuse",
       "point_lights[1].specular"

      },
      {std::make_tuple(particle_x->vbo_, 3, 0),
       std::make_tuple(particle_normalized_attr->vbo_, 1, 0)},
      [&](ShadingProgram& program, Display& display) {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glUniformMatrix4fv(
            program.get_uniform_location("P"), 1, GL_FALSE,
            glm::value_ptr(display.camera_.getProjectionMatrix()));
        glUniformMatrix4fv(program.get_uniform_location("V"), 1, GL_FALSE,
                           glm::value_ptr(display.camera_.getViewMatrix()));
        glUniform2f(program.get_uniform_location("screen_dimension"),
                    static_cast<GLfloat>(display.width_),
                    static_cast<GLfloat>(display.height_));
        glUniform1f(program.get_uniform_location("particle_radius"),
                    particle_radius);

        glm::vec3 const& camera_worldspace = display.camera_.getCenter();
        glUniform3f(program.get_uniform_location("camera_worldspace"),
                    camera_worldspace[0], camera_worldspace[1],
                    camera_worldspace[2]);
        glUniform3f(program.get_uniform_location("directional_light.direction"),
                    0.2f, 1.0f, 0.3f);
        glUniform3f(program.get_uniform_location("directional_light.ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("directional_light.diffuse"),
                    0.4f, 0.4f, 0.4f);
        glUniform3f(program.get_uniform_location("directional_light.specular"),
                    0.5f, 0.5f, 0.5f);

        glUniform3f(program.get_uniform_location("point_lights[0].position"),
                    2.0f, 2.0f, 2.0f);
        glUniform1f(program.get_uniform_location("point_lights[0].constant"),
                    1.0f);
        glUniform1f(program.get_uniform_location("point_lights[0].linear"),
                    0.09f);
        glUniform1f(program.get_uniform_location("point_lights[0].quadratic"),
                    0.032f);
        glUniform3f(program.get_uniform_location("point_lights[0].ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("point_lights[0].diffuse"),
                    0.8f, 0.8f, 0.8f);
        glUniform3f(program.get_uniform_location("point_lights[0].specular"),
                    1.0f, 1.0f, 1.0f);

        glUniform3f(program.get_uniform_location("point_lights[1].position"),
                    2.0f, 1.0f, -2.0f);
        glUniform1f(program.get_uniform_location("point_lights[1].constant"),
                    1.0f);
        glUniform1f(program.get_uniform_location("point_lights[1].linear"),
                    0.09f);
        glUniform1f(program.get_uniform_location("point_lights[1].quadratic"),
                    0.032f);
        glUniform3f(program.get_uniform_location("point_lights[1].ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("point_lights[1].diffuse"),
                    0.8f, 0.8f, 0.8f);
        glUniform3f(program.get_uniform_location("point_lights[1].specular"),
                    1.0f, 1.0f, 1.0f);
        glUniform3f(program.get_uniform_location("material.specular"), 0.8f,
                    0.9f, 0.9f);
        glUniform1f(program.get_uniform_location("material.shininess"), 5.0f);

        glBindTexture(GL_TEXTURE_1D, colormap_tex);
        for (I i = 0; i <= 0; ++i) {
          float wrap_length = grid_res.y * kernel_radius;
          glUniformMatrix4fv(
              program.get_uniform_location("M"), 1, GL_FALSE,
              glm::value_ptr(glm::translate(glm::mat4(1),
                                            glm::vec3{wrap_length * i, 0, 0})));
          glDrawArrays(GL_POINTS, 0, solver_df.num_particles);
        }
      }));

#include "alluvion/glsl/glyph.frag"
#include "alluvion/glsl/glyph.vert"
  display->add_shading_program(new ShadingProgram(
      kGlyphVertexShaderStr, kGlyphFragmentShaderStr,
      {
          "projection",
          "text_color",
      },
      {std::make_tuple(glyph_quad, 4, 0)},
      [&](ShadingProgram& program, Display& display) {
        glm::mat4 projection =
            glm::ortho(0.0f, static_cast<GLfloat>(display.width_), 0.0f,
                       static_cast<GLfloat>(display.height_));
        glUniformMatrix4fv(program.get_uniform_location("projection"), 1,
                           GL_FALSE, glm::value_ptr(projection));
        glUniform3f(program.get_uniform_location("text_color"), 1.0f, 1.0f,
                    1.0f);

        std::stringstream time_text;
        time_text << "num_particles = " << solver_df.num_particles << "("
                  << std::fixed << std::setprecision(3) << std::setw(5)
                  << naive_filled_percentage << " t: " << std::fixed
                  << std::setprecision(3) << std::setw(6) << t
                  << " dt: " << std::scientific << std::setprecision(3)
                  << std::setw(6) << solver_df.dt << " d: (" << std::scientific
                  << std::setprecision(3) << std::setw(6) << min_density_error
                  << "," << std::scientific << std::setprecision(3)
                  << std::setw(6) << max_density_error << ") v=("
                  << std::setw(6) << min_particle_speed << ","
                  << max_particle_speed << ")";
        std::string text = time_text.str();

        typesetter.start(display.width_ * 0.02f, display.height_ * 0.05f, 1.0f);
        for (std::string::const_iterator c = text.begin(); c != text.end();
             c++) {
          glBindTexture(GL_TEXTURE_2D, typesetter.place_glyph(*c));
          glBindBuffer(GL_ARRAY_BUFFER, glyph_quad);
          glBufferSubData(GL_ARRAY_BUFFER, 0, sizeof(typesetter.vertices_info_),
                          typesetter.vertices_info_);
          glBindBuffer(GL_ARRAY_BUFFER, 0);
          glDrawArrays(GL_TRIANGLES, 0, 6);
        }
        glBindTexture(GL_TEXTURE_2D, 0);
      }));
  display->run();
  // }}}
}
