#include "hip/hip_runtime.h"
#include <glm/gtc/type_ptr.hpp>
#include <iostream>
#include <limits>

#include "alluvion/colormaps.hpp"
#include "alluvion/constants.hpp"
#include "alluvion/dg/infinite_cylinder_distance.hpp"
#include "alluvion/dg/sphere_distance.hpp"
#include "alluvion/float_shorthands.hpp"
#include "alluvion/pile.hpp"
#include "alluvion/runner.hpp"
#include "alluvion/solver_df.hpp"
#include "alluvion/store.hpp"
#include "alluvion/typesetter.hpp"

using namespace alluvion;
using namespace alluvion::dg;

int main(void) {
  Store store;
  Display* display = store.create_display(800, 600, "particle view");
  Runner runner;

  F particle_radius = 0.0025_F;
  F kernel_radius = particle_radius * 4.0_F;
  F density0 = 1000.0_F;
  F cubical_particle_volume =
      8 * particle_radius * particle_radius * particle_radius;
  F volume_relative_to_cube = 0.8_F;
  F particle_mass =
      cubical_particle_volume * volume_relative_to_cube * density0;

  F3 gravity = {0._F, -9.81_F, 0._F};

  store.get_cn<F>().set_cubic_discretization_constants();
  store.get_cn<F>().set_kernel_radius(kernel_radius);
  store.get_cn<F>().set_particle_attr(particle_radius, particle_mass, density0);
  store.get_cn<F>().set_gravity(gravity);
  store.get_cn<F>().set_boundary_epsilon(1e-9_F);
  F target_physical_viscosity = 1e-3_F;
  F viscosity = 5e-6_F;
  F vorticity = 0.01_F;
  F inertia_inverse = 0.1_F;
  F viscosity_omega = 0.5_F;
  F surface_tension_coeff = 0.05_F;
  F surface_tension_boundary_coeff = 0.01_F;
  store.get_cn<F>().set_advanced_fluid_attr(
      viscosity, vorticity, inertia_inverse, viscosity_omega,
      surface_tension_coeff, surface_tension_boundary_coeff);

  I kM = 2;
  F cylinder_length = 2._F * kM * kernel_radius;
  I kQ = 5;
  F R = kernel_radius * kQ;

  const char* font_filename = "/usr/share/fonts/truetype/freefont/FreeMono.ttf";
  Typesetter typesetter(display, font_filename, 0, 30);
  typesetter.load_ascii();

  display->camera_.setEye(0._F, 0._F, R * 6._F);
  display->camera_.setClipPlanes(particle_radius * 10._F, R * 20._F);
  display->update_trackball_camera();

  GLuint colormap_tex =
      display->create_colormap(kViridisData.data(), kViridisData.size());

  GLuint glyph_quad = display->create_dynamic_array_buffer<float4>(6, nullptr);

  // rigids
  F restitution = 1._F;
  F friction = 0._F;
  F boundary_viscosity = viscosity * 1.5_F;
  U max_num_contacts = 512;
  Pile<F3, Q, F> pile(store, max_num_contacts);
  pile.add(new InfiniteCylinderDistance<F3, F>(R), U3{1, 20, 20}, -1._F, 0,
           Mesh(), 0._F, restitution, friction, boundary_viscosity, F3{1, 1, 1},
           F3{0, 0, 0}, Q{0, 0, 0, 1}, Mesh());
  pile.build_grids(4 * kernel_radius);
  pile.reallocate_kinematics_on_device();
  store.get_cni().set_num_boundaries(pile.get_size());
  store.get_cn<F>().set_contact_tolerance(0.05_F);

  // particles
  U max_num_particles =
      static_cast<U>(kPi<F> * (R - particle_radius) * (R - particle_radius) *
                     cylinder_length * density0 / particle_mass);

  // grid
  U3 grid_res{static_cast<U>(kM * 2), static_cast<U>(kQ * 2),
              static_cast<U>(kQ * 2)};
  I3 grid_offset{-kM, -kQ, -kQ};
  U max_num_particles_per_cell = 64;
  U max_num_neighbors_per_particle = 64;
  store.get_cni().init_grid_constants(grid_res, grid_offset);
  store.get_cni().set_max_num_particles_per_cell(max_num_particles_per_cell);
  store.get_cni().set_max_num_neighbors_per_particle(
      max_num_neighbors_per_particle);
  store.get_cn<F>().set_wrap_length(grid_res.x * kernel_radius);

  std::unique_ptr<GraphicalVariable<1, F3>> particle_x(
      store.create_graphical<1, F3>({max_num_particles}));
  std::unique_ptr<GraphicalVariable<1, F>> particle_normalized_attr(
      store.create_graphical<1, F>({max_num_particles}));
  Variable<1, F3> particle_v = store.create<1, F3>({max_num_particles});
  Variable<1, F3> particle_a = store.create<1, F3>({max_num_particles});
  Variable<1, F> particle_density = store.create<1, F>({max_num_particles});
  Variable<2, F3> particle_boundary_xj =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<2, F> particle_boundary_volume =
      store.create<2, F>({pile.get_size(), max_num_particles});
  Variable<2, F3> particle_force =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<2, F3> particle_torque =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<1, F> particle_cfl_v2 = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_dfsph_factor =
      store.create<1, F>({max_num_particles});
  Variable<1, F> particle_kappa = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_kappa_v = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_density_adv = store.create<1, F>({max_num_particles});
  Variable<4, Q> pid = store.create<4, Q>(
      {grid_res.x, grid_res.y, grid_res.z, max_num_particles_per_cell});
  Variable<3, U> pid_length =
      store.create<3, U>({grid_res.x, grid_res.y, grid_res.z});
  Variable<2, Q> particle_neighbors =
      store.create<2, Q>({max_num_particles, max_num_neighbors_per_particle});
  Variable<1, U> particle_num_neighbors =
      store.create<1, U>({max_num_particles});

  SolverDf<F3, Q, F> solver_df(
      runner, pile, *particle_x, *particle_normalized_attr, particle_v,
      particle_a, particle_density, particle_boundary_xj,
      particle_boundary_volume, particle_force, particle_torque,
      particle_cfl_v2, particle_dfsph_factor, particle_kappa, particle_kappa_v,
      particle_density_adv, pid, pid_length, particle_neighbors,
      particle_num_neighbors);
  solver_df.dt = 1e-3_F;
  solver_df.max_dt = 1e-5_F;
  solver_df.min_dt = 0.0_F;
  solver_df.cfl = 6e-3_F;
  solver_df.particle_radius = particle_radius;

  U initial_num_particles = 2700;
  F slice_distance = particle_radius * 2._F;
  U num_slices = static_cast<U>(cylinder_length / slice_distance);
  U num_particles_per_slice = initial_num_particles / num_slices;
  initial_num_particles = num_particles_per_slice * num_slices;

  store.copy_cn<F>();
  store.map_graphical_pointers();
  Runner::launch(initial_num_particles, 256, [&](U grid_size, U block_size) {
    create_fluid_cylinder<F3, F><<<grid_size, block_size>>>(
        *particle_x, initial_num_particles, R - particle_radius * 2._F,
        num_particles_per_slice, particle_radius * 2._F,
        cylinder_length * -0.5_F);
  });
  store.unmap_graphical_pointers();
  solver_df.num_particles = initial_num_particles;

  // sample points
  U num_sample_planes = 14;
  U num_samples_per_plane = 31;
  U num_samples = num_samples_per_plane * num_sample_planes;
  Variable<1, F3> sample_x = store.create<1, F3>({num_samples});
  Variable<1, F> sample_data = store.create<1, F>({num_samples});
  Variable<2, Q> sample_neighbors =
      store.create<2, Q>({num_samples, max_num_neighbors_per_particle});
  Variable<1, U> sample_num_neighbors = store.create<1, U>({num_samples});
  Variable<2, F3> sample_boundary_xj =
      store.create<2, F3>({pile.get_size(), num_samples});
  Variable<2, F> sample_boundary_volume =
      store.create<2, F>({pile.get_size(), num_samples});
  {
    std::vector<F3> sample_x_host(num_samples);
    F distance_between_sample_planes = cylinder_length / num_sample_planes;
    for (I i = 0; i < num_samples; ++i) {
      I plane_id = i / num_samples_per_plane;
      I id_in_plane = i % num_samples_per_plane;
      sample_x_host[i] = F3{
          cylinder_length * -0.5_F + distance_between_sample_planes * plane_id,
          R * 2._F / (num_samples_per_plane + 1) *
              (id_in_plane - static_cast<I>(num_samples_per_plane) / 2),
          0._F};
    }
    sample_x.set_bytes(sample_x_host.data());
  }
  std::vector<F> sample_data_host(num_samples);

  U step_id = 0;
  F t = 0;
  U num_emission = 1;
  // U num_emission = max_num_particles * 1 / 20;
  // U num_emission_slices = static_cast<U>(cylinder_length / slice_distance);
  // U num_emission_per_slice = num_emission / num_emission_slices;
  // num_emission = num_emission_per_slice * num_emission_slices;

  F next_emission_t = 0;
  // density sampling for the place with emission
  Variable<1, F3> emission_x = store.create<1, F3>({num_emission + 1});
  Variable<1, U> num_emitted = store.create<1, U>({1});
  Variable<1, F> emission_sample_density =
      store.create<1, F>({num_emission + 1});
  Variable<2, U> emission_neighbors =
      store.create<2, U>({num_emission + 1, max_num_neighbors_per_particle});
  Variable<1, U> emission_num_neighbors =
      store.create<1, U>({num_emission + 1});
  {
    std::vector<F3> emission_x_host(num_emission + 1);
    F pattern_radius = R - particle_radius * 2._F;
    F x_min = cylinder_length * -0.5_F;
    for (I i = 0; i < num_emission; ++i) {
      emission_x_host[i] =
          F3{cylinder_length * -0.5_F + i * particle_radius * 2._F,
             R - particle_radius * 2._F, 0._F};
    }
    emission_x_host[num_emission] = F3{0._F, -R + particle_radius * 2._F, 0._F};
    emission_x.set_bytes(emission_x_host.data());
  }
  std::vector<F> emission_sample_density_host(num_emission + 1);
  bool finished_filling = false;
  I consecutive_density_uniformity = -1;
  F max_density_error = std::numeric_limits<F>::max();
  F min_density_error = std::numeric_limits<F>::max();
  F naive_filled_percentage = 0;
  F finished_filling_t = -1.0_F;
  bool should_close = false;
  I resting_phase = 0;

  F max_particle_speed = 99.9_F;
  F min_particle_speed = 99.9_F;
  F sum_particle_velocity_components = 99.9_F;
  F last_stationary_t = 0._F;
  F last_emission_t = 0._F;
  bool emitted_since_last_stationary = true;

  display->add_shading_program(new ShadingProgram(
      nullptr, nullptr, {}, {}, [&](ShadingProgram& program, Display& display) {
        if (should_close) {
          return;
        }

        store.map_graphical_pointers();
        for (U frame_interstep = 0; frame_interstep < 10; ++frame_interstep) {
          // if (!finished_filling || step_id % 4000 == 0) {
          //   pid_length.set_zero();
          //   Runner::launch(
          //       solver_df.num_particles, 256, [&](U grid_size, U block_size)
          //       {
          //         update_particle_grid<<<grid_size, block_size>>>(
          //             *particle_x, pid, pid_length, solver_df.num_particles);
          //       });
          //   Runner::launch(num_samples, 256, [&](U grid_size, U block_size) {
          //     make_neighbor_list<1><<<grid_size, block_size>>>(
          //         sample_x,  pid, pid_length, sample_neighbors,
          //         sample_num_neighbors, num_samples);
          //   });
          //   Runner::launch(
          //       solver_df.num_particles, 256, [&](U grid_size, U block_size)
          //       {
          //         compute_density<<<grid_size, block_size>>>(
          //             *particle_x, particle_neighbors,
          //             particle_num_neighbors, particle_density,
          //             particle_boundary_xj, particle_boundary_volume,
          //             solver_df.num_particles);
          //       });
          //   Runner::launch(num_samples, 256, [&](U grid_size, U block_size) {
          //     sample_fluid<<<grid_size, block_size>>>(
          //         sample_x, *particle_x, particle_density, particle_density,
          //         sample_neighbors, sample_num_neighbors, sample_data,
          //         num_samples);
          //   });
          //   Runner::launch(
          //       num_emission + 1, 256, [&](U grid_size, U block_size) {
          //         make_neighbor_list<1><<<grid_size, block_size>>>(
          //             emission_x,  pid, pid_length,
          //             emission_neighbors, emission_num_neighbors,
          //             num_emission + 1);
          //       });
          //   Runner::launch(num_emission + 1, 256,
          //                  [&](U grid_size, U block_size) {
          //                    sample_fluid<<<grid_size, block_size>>>(
          //                        emission_x, *particle_x, particle_density,
          //                        particle_density, emission_neighbors,
          //                        emission_num_neighbors,
          //                        emission_sample_density, num_emission + 1);
          //                  });
          //   sample_data.get_bytes(
          //       sample_data_host.data());  // for determining whether top
          //                                  // density matches bottom density
          // }
          // if (finished_filling && t - finished_filling_t > 3._F &&
          //     resting_phase == 2) {
          //   resting_phase++;
          //   particle_x->write_file("x5-revamp.alu", solver_df.num_particles);
          //   particle_v.write_file("v5-revamp.alu", solver_df.num_particles);
          //   std::cout << "Finished resting phase 3." << std::endl;
          // } else if (finished_filling && t - finished_filling_t > 4._F &&
          //            resting_phase == 1 && max_particle_speed < 1e-6_F) {
          //   resting_phase++;
          //   std::cout << "Finished resting phase 2." << std::endl;
          // } else if (finished_filling && t - finished_filling_t > 4._F &&
          //            resting_phase == 0 && max_particle_speed < 1e-5_F) {
          //   resting_phase++;
          //   std::cout << "Finished resting phase 1." << std::endl;
          // }
          // if (finished_filling && t - finished_filling_t < 3.0_F) {
          //   if (step_id % 1000 == 0) {
          //     particle_v.set_zero();
          //   }
          // }

          // ===== [emission
          F3 emission_velocity = gravity * solver_df.dt * 500.0_F;
          F emission_speed = length(emission_velocity);
          F emission_interval = particle_radius * 2._F / emission_speed;
          if (solver_df.num_particles < 2704 && !finished_filling &&
              max_density_error < 1.5e-4_F && t >= next_emission_t &&
              t - last_emission_t > 0.5_F &&
              consecutive_density_uniformity < 0) {
            num_emitted.set_zero();
            Runner::launch(num_emission, 256, [&](U grid_size, U block_size) {
              emit_if_density_lower_than_last<<<grid_size, block_size>>>(
                  *particle_x, particle_v, emission_x, emission_sample_density,
                  num_emitted, num_emission, solver_df.num_particles, 0.99_F,
                  emission_velocity);
            });
            U num_emitted_host;
            num_emitted.get_bytes(&num_emitted_host);
            solver_df.num_particles += num_emitted_host;
            std::cout << "filled " << num_emitted_host << std::endl;
            last_emission_t = t;
            if (num_emitted_host > 0) {
              next_emission_t = t + emission_interval;
              emitted_since_last_stationary = true;
            }
            if (solver_df.num_particles == 2704) {
              std::cout << "set gravity to zero" << std::endl;
              store.get_cn<F>().set_gravity(F3{});
            }
          }
          // ===== ]emission
          if (t > 10._F && solver_df.num_particles == 2704 &&
              t - last_stationary_t > 0.2_F) {
            particle_x->write_file("x5-2704.alu", solver_df.num_particles);
            should_close = true;
          }
          if (min_particle_speed > 2._F || (step_id % 1000 == 0)) {
            particle_v.set_zero();
            particle_dfsph_factor.set_zero();
            particle_kappa.set_zero();
            particle_kappa_v.set_zero();
            particle_density_adv.set_zero();
            last_stationary_t = t;
            emitted_since_last_stationary = false;
            std::cout << "last stationary t = " << last_stationary_t
                      << std::endl;
          }

          solver_df.step<1>();

          t += solver_df.dt;
          step_id += 1;

          max_density_error =
              Runner::max(particle_density, solver_df.num_particles) /
                  density0 -
              1;
          min_density_error =
              Runner::min(particle_density, solver_df.num_particles) /
                  density0 -
              1;
          max_particle_speed =
              sqrt(Runner::max(particle_cfl_v2, solver_df.num_particles));
          min_particle_speed =
              sqrt(Runner::min(particle_cfl_v2, solver_df.num_particles));
          sum_particle_velocity_components =
              Runner::sum<F>(particle_v.ptr_, particle_v.get_num_primitives());
          F expected_total_volume = kPi<F> * (R - particle_radius) *
                                    (R - particle_radius) * cylinder_length;
          naive_filled_percentage = solver_df.num_particles * particle_mass /
                                    density0 / expected_total_volume;
          // F lower_density_ratio = min_density / density0;
          // if (lower_density_ratio > 0.98) {
          //   num_emission = 1;
          //   {
          //     std::vector<F3> emission_x_host(num_emission + 1);
          //     F pattern_radius = R - particle_radius * 2._F;
          //     F x_min = cylinder_length * -0.5_F;
          //     for (I i = 0; i < num_emission; ++i) {
          //       emission_x_host[i] =
          //           F3{cylinder_length * -0.5_F + i * particle_radius * 2._F,
          //              R - particle_radius * 2._F, 0._F};
          //     }
          //     emission_x_host[num_emission] =
          //         F3{0._F, -R + particle_radius * 2._F, 0._F};
          //     emission_x.set_bytes(emission_x_host.data());
          //   }
          // } else if (lower_density_ratio > 0.95) {
          //   num_emission = 2;
          //   {
          //     std::vector<F3> emission_x_host(num_emission + 1);
          //     F pattern_radius = R - particle_radius * 2._F;
          //     F x_min = cylinder_length * -0.5_F;
          //     for (I i = 0; i < num_emission; ++i) {
          //       emission_x_host[i] =
          //           F3{cylinder_length * -0.5_F + i * particle_radius * 2._F,
          //              R - particle_radius * 2._F, 0._F};
          //     }
          //     emission_x_host[num_emission] =
          //         F3{0._F, -R + particle_radius * 2._F, 0._F};
          //     emission_x.set_bytes(emission_x_host.data());
          //   }
          // } else if (lower_density_ratio > 0.90) {
          //   num_emission = 4;
          //   {
          //     std::vector<F3> emission_x_host(num_emission + 1);
          //     F pattern_radius = R - particle_radius * 2._F;
          //     F x_min = cylinder_length * -0.5_F;
          //     for (I i = 0; i < num_emission; ++i) {
          //       emission_x_host[i] =
          //           F3{cylinder_length * -0.5_F + i * particle_radius * 2._F,
          //              R - particle_radius * 2._F, 0._F};
          //     }
          //     emission_x_host[num_emission] =
          //         F3{0._F, -R + particle_radius * 2._F, 0._F};
          //     emission_x.set_bytes(emission_x_host.data());
          //   }
          // }
          // if ((max_density - min_density) / density0 < 5e-3_F) {
          //   ++consecutive_density_uniformity;
          //   F density_diff =
          //       abs(sample_data_host[0] - sample_data_host[num_samples - 1])
          //       / density0;
          //   if (density_diff < 0.005_F && consecutive_density_uniformity > 50
          //   &&
          //       !finished_filling) {
          //     finished_filling = true;
          //     store.get_cn<F>().set_gravity(F3{});
          //     finished_filling_t = t;
          //     std::cout << "finished filling" << std::endl;
          //   }
          // } else {
          //   consecutive_density_uniformity = -1;
          // }

          // if (step_id % 20000 == 0) {
          //   std::cout << "particle speed = " << min_particle_speed << ", "
          //             << max_particle_speed << " sum of components = "
          //             << sum_particle_velocity_components << std::endl;
          // }
        }
        solver_df.colorize_speed(0, 2.0);
        store.unmap_graphical_pointers();
      }));

  // {{{
#include "alluvion/glsl/particle.frag"
#include "alluvion/glsl/particle.vert"
  display->add_shading_program(new ShadingProgram(
      kParticleVertexShaderStr, kParticleFragmentShaderStr,
      {"particle_radius", "screen_dimension", "M", "V", "P",
       "camera_worldspace", "material.specular", "material.shininess",
       "directional_light.direction", "directional_light.ambient",
       "directional_light.diffuse", "directional_light.specular",
       "point_lights[0].position", "point_lights[0].constant",
       "point_lights[0].linear", "point_lights[0].quadratic",
       "point_lights[0].ambient", "point_lights[0].diffuse",
       "point_lights[0].specular",
       //
       "point_lights[1].position", "point_lights[1].constant",
       "point_lights[1].linear", "point_lights[1].quadratic",
       "point_lights[1].ambient", "point_lights[1].diffuse",
       "point_lights[1].specular"

      },
      {std::make_tuple(particle_x->vbo_, 3, 0),
       std::make_tuple(particle_normalized_attr->vbo_, 1, 0)},
      [&](ShadingProgram& program, Display& display) {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glUniformMatrix4fv(
            program.get_uniform_location("P"), 1, GL_FALSE,
            glm::value_ptr(display.camera_.getProjectionMatrix()));
        glUniformMatrix4fv(program.get_uniform_location("V"), 1, GL_FALSE,
                           glm::value_ptr(display.camera_.getViewMatrix()));
        glUniform2f(program.get_uniform_location("screen_dimension"),
                    static_cast<GLfloat>(display.width_),
                    static_cast<GLfloat>(display.height_));
        glUniform1f(program.get_uniform_location("particle_radius"),
                    particle_radius);

        glm::vec3 const& camera_worldspace = display.camera_.getCenter();
        glUniform3f(program.get_uniform_location("camera_worldspace"),
                    camera_worldspace[0], camera_worldspace[1],
                    camera_worldspace[2]);
        glUniform3f(program.get_uniform_location("directional_light.direction"),
                    0.2f, 1.0f, 0.3f);
        glUniform3f(program.get_uniform_location("directional_light.ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("directional_light.diffuse"),
                    0.4f, 0.4f, 0.4f);
        glUniform3f(program.get_uniform_location("directional_light.specular"),
                    0.5f, 0.5f, 0.5f);

        glUniform3f(program.get_uniform_location("point_lights[0].position"),
                    2.0f, 2.0f, 2.0f);
        glUniform1f(program.get_uniform_location("point_lights[0].constant"),
                    1.0f);
        glUniform1f(program.get_uniform_location("point_lights[0].linear"),
                    0.09f);
        glUniform1f(program.get_uniform_location("point_lights[0].quadratic"),
                    0.032f);
        glUniform3f(program.get_uniform_location("point_lights[0].ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("point_lights[0].diffuse"),
                    0.8f, 0.8f, 0.8f);
        glUniform3f(program.get_uniform_location("point_lights[0].specular"),
                    1.0f, 1.0f, 1.0f);

        glUniform3f(program.get_uniform_location("point_lights[1].position"),
                    2.0f, 1.0f, -2.0f);
        glUniform1f(program.get_uniform_location("point_lights[1].constant"),
                    1.0f);
        glUniform1f(program.get_uniform_location("point_lights[1].linear"),
                    0.09f);
        glUniform1f(program.get_uniform_location("point_lights[1].quadratic"),
                    0.032f);
        glUniform3f(program.get_uniform_location("point_lights[1].ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("point_lights[1].diffuse"),
                    0.8f, 0.8f, 0.8f);
        glUniform3f(program.get_uniform_location("point_lights[1].specular"),
                    1.0f, 1.0f, 1.0f);
        glUniform3f(program.get_uniform_location("material.specular"), 0.8f,
                    0.9f, 0.9f);
        glUniform1f(program.get_uniform_location("material.shininess"), 5.0f);

        glBindTexture(GL_TEXTURE_1D, colormap_tex);
        for (I i = 0; i <= 0; ++i) {
          float wrap_length = grid_res.x * kernel_radius;
          glUniformMatrix4fv(
              program.get_uniform_location("M"), 1, GL_FALSE,
              glm::value_ptr(glm::translate(glm::mat4(1),
                                            glm::vec3{wrap_length * i, 0, 0})));
          glDrawArrays(GL_POINTS, 0, solver_df.num_particles);
        }
      }));

#include "alluvion/glsl/glyph.frag"
#include "alluvion/glsl/glyph.vert"
  display->add_shading_program(new ShadingProgram(
      kGlyphVertexShaderStr, kGlyphFragmentShaderStr,
      {
          "projection",
          "text_color",
      },
      {std::make_tuple(glyph_quad, 4, 0)},
      [&](ShadingProgram& program, Display& display) {
        glm::mat4 projection =
            glm::ortho(0.0f, static_cast<GLfloat>(display.width_), 0.0f,
                       static_cast<GLfloat>(display.height_));
        glUniformMatrix4fv(program.get_uniform_location("projection"), 1,
                           GL_FALSE, glm::value_ptr(projection));
        glUniform3f(program.get_uniform_location("text_color"), 1.0f, 1.0f,
                    1.0f);

        std::stringstream time_text;
        time_text << "num_particles = " << solver_df.num_particles << "("
                  << std::fixed << std::setprecision(3) << std::setw(5)
                  << naive_filled_percentage << " t: " << std::fixed
                  << std::setprecision(3) << std::setw(6) << t
                  << " dt: " << std::scientific << std::setprecision(3)
                  << std::setw(6) << solver_df.dt << " d: (" << std::scientific
                  << std::setprecision(3) << std::setw(6) << min_density_error
                  << "," << std::scientific << std::setprecision(3)
                  << std::setw(6) << max_density_error << ") v=("
                  << std::setw(6) << min_particle_speed << ","
                  << max_particle_speed << ")";
        std::string text = time_text.str();

        typesetter.start(display.width_ * 0.1f, display.height_ * 0.05f, 1.0f);
        for (std::string::const_iterator c = text.begin(); c != text.end();
             c++) {
          glBindTexture(GL_TEXTURE_2D, typesetter.place_glyph(*c));
          glBindBuffer(GL_ARRAY_BUFFER, glyph_quad);
          glBufferSubData(GL_ARRAY_BUFFER, 0, sizeof(typesetter.vertices_info_),
                          typesetter.vertices_info_);
          glBindBuffer(GL_ARRAY_BUFFER, 0);
          glDrawArrays(GL_TRIANGLES, 0, 6);
        }
        glBindTexture(GL_TEXTURE_2D, 0);
      }));
  display->run();
  // }}}
}
