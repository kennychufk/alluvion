#include "hip/hip_runtime.h"
#include <glm/gtc/type_ptr.hpp>
#include <iostream>

#include "alluvion/constants.hpp"
#include "alluvion/dg/sphere_distance.hpp"
#include "alluvion/pile.hpp"
#include "alluvion/runner.hpp"
#include "alluvion/store.hpp"

using namespace alluvion;
using namespace alluvion::dg;

int main(void) {
  Store store;
  Display* display = store.create_display(800, 600, "particle view");

  F particle_radius = 0.25;
  F kernel_radius = 1.0;
  F density0 = 1.0;
  F particle_mass = 0.1;
  F dt = 1e-3;
  F gravity = -9.81;
  cnst::set_cubic_discretization_constants();
  cnst::set_kernel_radius(kernel_radius);
  cnst::set_particle_attr(particle_radius, particle_mass, density0);
  cnst::set_gravity(gravity);

  U num_particles = 10000;
  U3 grid_res{128, 128, 128};
  U num_grid_cells = grid_res.x * grid_res.y * grid_res.z;
  I3 grid_offset{-64, -64, -64};
  U max_num_particles_per_cell = 128;
  U max_num_neighbors_per_particle = 128;
  const F kCellWidthRelativeToKernelRadius =
      pow((sqrt(5.0) - 1.0) * 0.5, 1.0 / 3.0);
  cnst::init_grid_constants(grid_res, grid_offset);
  cnst::set_cell_width(kernel_radius * kCellWidthRelativeToKernelRadius);
  cnst::set_search_range(2.0 / kCellWidthRelativeToKernelRadius);

  cnst::set_max_num_particles_per_cell(max_num_particles_per_cell);
  cnst::set_max_num_neighbors_per_particle(max_num_neighbors_per_particle);

  // rigids
  Pile pile(store);
  pile.add("cube.obj", U3{50, 50, 50}, -1.0_F, 0, nullptr, 1, 1, 0, 0.2,
           F3{1, 1, 1}, F3{0, 0, 0}, Q{0, 0, 0, 1}, nullptr);
  pile.add(new SphereDistance(3.0_F), U3{50, 50, 50}, 1.0_F, 0, {}, 1, 1, 0,
           0.2, F3{1, 1, 1}, F3{-6, -6, -6}, Q{0, 0, 0, 1}, {}, {});
  pile.build_grids(4 * kernel_radius);
  pile.reallocate_kinematics_on_device();

  // particles
  GraphicalVariable<1, F3> particle_x =
      store.create_graphical<1, F3>({num_particles});
  Variable<1, F3> particle_v = store.create<1, F3>({num_particles});
  Variable<1, F3> particle_a = store.create<1, F3>({num_particles});
  Variable<1, F> particle_density = store.create<1, F>({num_particles});
  Variable<1, F> particle_pressure = store.create<1, F>({num_particles});
  Variable<1, F> particle_last_pressure = store.create<1, F>({num_particles});
  Variable<2, F3> particle_boundary_xj =
      store.create<2, F3>({pile.get_size(), num_particles});
  Variable<2, F> particle_boundary_volume =
      store.create<2, F>({pile.get_size(), num_particles});
  Variable<2, F3> particle_force =
      store.create<2, F3>({pile.get_size(), num_particles});
  Variable<2, F3> particle_torque =
      store.create<2, F3>({pile.get_size(), num_particles});
  Variable<1, F> particle_aii = store.create<1, F>({num_particles});
  Variable<1, F3> particle_dii = store.create<1, F3>({num_particles});
  Variable<1, F3> particle_dij_pj = store.create<1, F3>({num_particles});
  Variable<1, F> particle_sum_tmp = store.create<1, F>({num_particles});
  Variable<1, F> particle_adv_density = store.create<1, F>({num_particles});
  Variable<1, F3> particle_pressure_accel =
      store.create<1, F3>({num_particles});

  // grid
  Variable<4, U> pid = store.create<4, U>(
      {grid_res.x, grid_res.y, grid_res.z, max_num_particles_per_cell});
  Variable<3, U> pid_length =
      store.create<3, U>({grid_res.x, grid_res.y, grid_res.z});
  // neighbor
  Variable<2, U> particle_neighbors =
      store.create<2, U>({num_particles, max_num_neighbors_per_particle});
  Variable<1, U> particle_num_neighbors = store.create<1, U>({num_particles});

  store.map_graphical_pointers();
  Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
    create_fluid_block<F3, F>
        <<<grid_size, block_size>>>(particle_x, num_particles, 0, 1,
                                    F3{-5.0, -5.0, -5.0}, F3{5.0, 5.0, 5.0});
  });

  store.unmap_graphical_pointers();

  U frame_id = 0;
  display->add_shading_program(new ShadingProgram(
      nullptr, nullptr, {}, [&](ShadingProgram& program, Display& display) {
        // std::cout << "============= frame_id = " << frame_id << std::endl;

        store.map_graphical_pointers();
        // start of simulation loop
        pile.copy_kinematics_to_device();
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          clear_acceleration<<<grid_size, block_size>>>(particle_a,
                                                        num_particles);
        });
        pile.for_each_rigid([&](U boundary_id,
                                Variable<1, F> const& distance_grid,
                                Variable<1, F> const& volume_grid,
                                F3 const& rigid_x, Q const& rigid_q,
                                F3 const& domain_min, F3 const& domain_max,
                                U3 const& resolution, F3 const& cell_size,
                                U num_nodes, F sign, F thickness) {
          Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
            compute_particle_boundary<<<grid_size, block_size>>>(
                volume_grid, distance_grid, rigid_x, rigid_q, boundary_id,
                domain_min, domain_max, resolution, cell_size, num_nodes, 0,
                sign, thickness, dt, particle_x, particle_v,
                particle_boundary_xj, particle_boundary_volume, num_particles);
          });
        });
        Runner::launch(num_grid_cells, 256, [&](U grid_size, U block_size) {
          clear_particle_grid<<<grid_size, block_size>>>(pid_length,
                                                         num_grid_cells);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          update_particle_grid<<<grid_size, block_size>>>(
              particle_x, pid, pid_length, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          make_neighbor_list<<<grid_size, block_size>>>(
              particle_x, pid, pid_length, particle_neighbors,
              particle_num_neighbors, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          compute_density_fluid<<<grid_size, block_size>>>(
              particle_x, particle_neighbors, particle_num_neighbors,
              particle_density, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          compute_density_boundary<<<grid_size, block_size>>>(
              particle_x, particle_density, particle_boundary_xj,
              particle_boundary_volume, num_particles);
        });
        // compute_normal
        // compute_surface_tension_fluid
        // compute_surface_tension_boundary

        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          compute_viscosity_fluid<<<grid_size, block_size>>>(
              particle_x, particle_v, particle_density, particle_neighbors,
              particle_num_neighbors, particle_a, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          compute_viscosity_boundary<<<grid_size, block_size>>>(
              particle_x, particle_v, particle_a, particle_force,
              particle_torque, particle_boundary_xj, particle_boundary_volume,
              pile.x_device_, pile.v_device_, pile.omega_device_,
              pile.boundary_viscosity_device_, num_particles);
        });

        // reset_angular_acceleration
        // compute_vorticity_fluid
        // compute_vorticity_boundary
        // integrate_angular_acceleration
        //
        // calculate_cfl_v2
        // update_dt

        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          predict_advection0_fluid_advect<<<grid_size, block_size>>>(
              particle_v, particle_a, dt, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          predict_advection0_fluid<<<grid_size, block_size>>>(
              particle_x, particle_density, particle_dii, particle_neighbors,
              particle_num_neighbors, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          predict_advection0_boundary<<<grid_size, block_size>>>(
              particle_x, particle_density, particle_dii, particle_boundary_xj,
              particle_boundary_volume, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          reset_last_pressure<<<grid_size, block_size>>>(
              particle_pressure, particle_last_pressure, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          predict_advection1_fluid<<<grid_size, block_size>>>(
              particle_x, particle_v, particle_dii, particle_adv_density,
              particle_aii, particle_density, particle_neighbors,
              particle_num_neighbors, dt, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          predict_advection1_boundary<<<grid_size, block_size>>>(
              particle_x, particle_v, particle_density, particle_dii,
              particle_adv_density, particle_aii, particle_boundary_xj,
              particle_boundary_volume, pile.x_device_, pile.v_device_,
              pile.omega_device_, dt, num_particles);
        });

        for (U p_solve_iteration = 0; p_solve_iteration < 5;
             ++p_solve_iteration) {
          Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
            pressure_solve_iteration0<<<grid_size, block_size>>>(
                particle_x, particle_density, particle_last_pressure,
                particle_dij_pj, particle_neighbors, particle_num_neighbors,
                num_particles);
          });
          Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
            pressure_solve_iteration1_fluid<<<grid_size, block_size>>>(
                particle_x, particle_density, particle_last_pressure,
                particle_dii, particle_dij_pj, particle_sum_tmp,
                particle_neighbors, particle_num_neighbors, num_particles);
          });
          Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
            pressure_solve_iteration1_boundary<<<grid_size, block_size>>>(
                particle_x, particle_dij_pj, particle_sum_tmp,
                particle_boundary_xj, particle_boundary_volume, num_particles);
          });
          Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
            pressure_solve_iteration1_summarize<<<grid_size, block_size>>>(
                particle_aii, particle_adv_density, particle_sum_tmp,
                particle_last_pressure, particle_pressure, dt, num_particles);
          });
        }

        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          compute_pressure_accels_fluid<<<grid_size, block_size>>>(
              particle_x, particle_density, particle_pressure,
              particle_pressure_accel, particle_neighbors,
              particle_num_neighbors, num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          compute_pressure_accels_boundary<<<grid_size, block_size>>>(
              particle_x, particle_density, particle_pressure,
              particle_pressure_accel, particle_force, particle_torque,
              particle_boundary_xj, particle_boundary_volume, pile.x_device_,
              num_particles);
        });
        Runner::launch(num_particles, 256, [&](U grid_size, U block_size) {
          kinematic_integration<<<grid_size, block_size>>>(
              particle_x, particle_v, particle_pressure_accel, dt,
              num_particles);
        });

        store.unmap_graphical_pointers();
        frame_id += 1;
      }));
  // {{{
  display->add_shading_program(new ShadingProgram(
      R"CODE(
#version 330 core
layout(location = 0) in vec3 x;
uniform mat4 view_matrix;
uniform mat4 clip_matrix;
uniform vec2 screen_dimension;
uniform float point_scale;

out vec3 eyePos;
out float eyeRadius;

void main() {
  vec4 camera_space_x4 = view_matrix * vec4(x, 1.0);

  eyePos = camera_space_x4.xyz;
  eyeRadius = point_scale / -camera_space_x4.z / screen_dimension.y;
  gl_Position = clip_matrix * camera_space_x4;
  gl_PointSize = point_scale / -camera_space_x4.z;
}
)CODE",
      R"CODE(
#version 330 core
uniform vec4 base_color;

out vec4 output_color;

void main() {
  const vec3 light_direction = vec3(0.577, 0.577, 0.577);

  vec3 N;
  N.xy = gl_PointCoord * vec2(2.0, -2.0) + vec2(-1.0, 1.0);
  float N_squared = dot(N.xy, N.xy);
  if (N_squared > 1.0) discard;
  N.z = sqrt(1.0 - N_squared);

  float diffuse = max(0.0, dot(light_direction, N));
  output_color = base_color * diffuse;
}
)CODE",
      {"view_matrix", "clip_matrix", "screen_dimension", "point_scale",
       "base_color"},
      [&particle_x, num_particles](ShadingProgram& program, Display& display) {
        glm::mat4 clip_matrix = glm::perspective(
            glm::radians(45.0f),
            display.width_ / static_cast<GLfloat>(display.height_), .01f,
            100.f);

        glUniformMatrix4fv(program.get_uniform_location("view_matrix"), 1,
                           GL_FALSE,
                           glm::value_ptr(display.camera_.getMatrix()));
        glUniformMatrix4fv(program.get_uniform_location("clip_matrix"), 1,
                           GL_FALSE, glm::value_ptr(clip_matrix));
        glUniform2f(program.get_uniform_location("screen_dimension"),
                    static_cast<GLfloat>(display.width_),
                    static_cast<GLfloat>(display.height_));
        glUniform1f(program.get_uniform_location("point_scale"), 400);
        glUniform4f(program.get_uniform_location("base_color"), 1.0, 1.0, 1.0,
                    1.0);

        glBindBuffer(GL_ARRAY_BUFFER, particle_x.vbo_);
        glEnableVertexAttribArray(0);
        glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);
        glDrawArrays(GL_POINTS, 0, num_particles);
        glDisableVertexAttribArray(0);
      }));

  // rigid mesh shader
  display->add_shading_program(new ShadingProgram(
      R"CODE(
#version 330 core
layout(location = 0) in vec3 x;
uniform mat4 model_matrix;
uniform mat4 view_matrix;
uniform mat4 clip_matrix;

void main() {
  gl_Position = clip_matrix * view_matrix * model_matrix * vec4(x, 1.0);
}
)CODE",
      R"CODE(
#version 330 core
uniform vec4 base_color;

out vec4 output_color;

void main() {
  output_color = base_color;
}
)CODE",
      {"model_matrix", "view_matrix", "clip_matrix", "base_color"},
      [&pile](ShadingProgram& program, Display& display) {
        glm::mat4 clip_matrix = glm::perspective(
            glm::radians(45.0f),
            display.width_ / static_cast<GLfloat>(display.height_), .01f,
            100.f);

        glm::mat4 model_matrix = pile.get_matrix(0);
        glUniformMatrix4fv(program.get_uniform_location("model_matrix"), 1,
                           GL_FALSE, glm::value_ptr(model_matrix));
        glUniformMatrix4fv(program.get_uniform_location("view_matrix"), 1,
                           GL_FALSE,
                           glm::value_ptr(display.camera_.getMatrix()));
        glUniformMatrix4fv(program.get_uniform_location("clip_matrix"), 1,
                           GL_FALSE, glm::value_ptr(clip_matrix));
        glUniform4f(program.get_uniform_location("base_color"), 0.9, 0.3, 0.4,
                    1.0);

        MeshBuffer const& mesh_buffer = pile.mesh_buffer_list_[0];
        glBindBuffer(GL_ARRAY_BUFFER, mesh_buffer.vertex);
        glEnableVertexAttribArray(0);
        glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);
        glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, mesh_buffer.index);
        glDrawElements(GL_TRIANGLES, mesh_buffer.num_indices, GL_UNSIGNED_INT,
                       0);
        glDisableVertexAttribArray(0);
      }));
  display->run();
  // }}}
}
