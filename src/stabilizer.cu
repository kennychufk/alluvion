#include "hip/hip_runtime.h"
#include <cassert>
#include <glm/gtc/type_ptr.hpp>
#include <iostream>
#include <limits>
#include <random>

#include "alluvion/colormaps.hpp"
#include "alluvion/constants.hpp"
#include "alluvion/dg/infinite_cylinder_distance.hpp"
#include "alluvion/dg/sphere_distance.hpp"
#include "alluvion/float_shorthands.hpp"
#include "alluvion/pile.hpp"
#include "alluvion/runner.hpp"
#include "alluvion/solver_df.hpp"
#include "alluvion/store.hpp"
#include "alluvion/typesetter.hpp"

using namespace alluvion;
using namespace alluvion::dg;

int main(int argc, char* argv[]) {
  Store store;
  Display* display = store.create_display(800, 600, "particle view");
  Runner runner;

  F particle_radius = 0.0025_F;
  F kernel_radius = particle_radius * 4.0_F;
  F density0 = 1000.0_F;
  F cubical_particle_volume =
      8 * particle_radius * particle_radius * particle_radius;
  F volume_relative_to_cube = 0.8_F;
  F particle_mass =
      cubical_particle_volume * volume_relative_to_cube * density0;

  store.get_cn<F>().set_cubic_discretization_constants();
  store.get_cn<F>().set_kernel_radius(kernel_radius);
  store.get_cn<F>().set_particle_attr(particle_radius, particle_mass, density0);
  store.get_cn<F>().gravity = F3{0, 0.0, 0};
  store.get_cn<F>().boundary_epsilon = 1e-9_F;
  F viscosity = 5e-6_F;
  F vorticity = 0.01_F;
  F inertia_inverse = 0.1_F;
  F viscosity_omega = 0.5_F;
  F surface_tension_coeff = 0.05_F;
  F surface_tension_boundary_coeff = 0.01_F;
  store.get_cn<F>().viscosity = viscosity;

  I kM = 2;
  F cylinder_length = 2._F * kM * kernel_radius;
  I kQ = 5;
  F R = kernel_radius * kQ;

  const char* font_filename = "/usr/share/fonts/truetype/freefont/FreeMono.ttf";
  Typesetter typesetter(display, font_filename, 0, 30);
  typesetter.load_ascii();

  display->camera_.setEye(0._F, 0._F, R * 6._F);
  display->camera_.setClipPlanes(particle_radius * 10._F, R * 20._F);
  display->update_trackball_camera();

  GLuint colormap_tex =
      display->create_colormap(kViridisData.data(), kViridisData.size());

  GLuint glyph_quad = display->create_dynamic_array_buffer<float4>(6, nullptr);

  // rigids
  F restitution = 1._F;
  F friction = 0._F;
  F boundary_viscosity = viscosity * 1.5_F;
  U max_num_contacts = 512;
  Pile<F3, Q, F> pile(store, max_num_contacts);
  pile.add(new InfiniteCylinderDistance<F3, F>(R), U3{64, 1, 64}, -1._F, 0,
           Mesh(), 0._F, restitution, friction, boundary_viscosity, F3{1, 1, 1},
           F3{0, 0, 0}, Q{0, 0, 0, 1}, Mesh());
  pile.build_grids(4 * kernel_radius);
  pile.reallocate_kinematics_on_device();
  store.get_cn<F>().contact_tolerance = particle_radius;

  // particles
  U max_num_particles = static_cast<U>(kPi<F> * R * R * cylinder_length *
                                       density0 / particle_mass);
  // grid
  U3 grid_res{static_cast<U>(kQ * 2), static_cast<U>(kM * 2),
              static_cast<U>(kQ * 2)};
  I3 grid_offset{-kQ, -kM, -kQ};
  U max_num_particles_per_cell = 64;
  U max_num_neighbors_per_particle = 64;
  store.get_cni().grid_res = grid_res;
  store.get_cni().grid_offset = grid_offset;
  store.get_cni().max_num_particles_per_cell = max_num_particles_per_cell;
  store.get_cni().max_num_neighbors_per_particle =
      max_num_neighbors_per_particle;
  store.get_cn<F>().set_wrap_length(grid_res.y * kernel_radius);

  std::unique_ptr<GraphicalVariable<1, F3>> particle_x(
      store.create_graphical<1, F3>({max_num_particles}));
  std::unique_ptr<GraphicalVariable<1, F>> particle_normalized_attr(
      store.create_graphical<1, F>({max_num_particles}));
  Variable<1, F3> particle_v = store.create<1, F3>({max_num_particles});
  Variable<1, F3> particle_a = store.create<1, F3>({max_num_particles});
  Variable<1, F> particle_density = store.create<1, F>({max_num_particles});
  Variable<2, F3> particle_boundary_xj =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<2, F> particle_boundary_volume =
      store.create<2, F>({pile.get_size(), max_num_particles});
  Variable<2, F3> particle_force =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<2, F3> particle_torque =
      store.create<2, F3>({pile.get_size(), max_num_particles});
  Variable<1, F> particle_cfl_v2 = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_dfsph_factor =
      store.create<1, F>({max_num_particles});
  Variable<1, F> particle_kappa = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_kappa_v = store.create<1, F>({max_num_particles});
  Variable<1, F> particle_density_adv = store.create<1, F>({max_num_particles});
  Variable<4, Q> pid = store.create<4, Q>(
      {grid_res.x, grid_res.y, grid_res.z, max_num_particles_per_cell});
  Variable<3, U> pid_length =
      store.create<3, U>({grid_res.x, grid_res.y, grid_res.z});
  Variable<2, Q> particle_neighbors =
      store.create<2, Q>({max_num_particles, max_num_neighbors_per_particle});
  Variable<1, U> particle_num_neighbors =
      store.create<1, U>({max_num_particles});

  SolverDf<F3, Q, F> solver_df(
      runner, pile, *particle_x, *particle_normalized_attr, particle_v,
      particle_a, particle_density, particle_boundary_xj,
      particle_boundary_volume, particle_force, particle_torque,
      particle_cfl_v2, particle_dfsph_factor, particle_kappa, particle_kappa_v,
      particle_density_adv, pid, pid_length, particle_neighbors,
      particle_num_neighbors);
  solver_df.dt = 1e-3_F;
  solver_df.max_dt = 1e-4_F;
  solver_df.min_dt = 0.0_F;
  solver_df.cfl = 2e-3_F;
  solver_df.particle_radius = particle_radius;

  store.copy_cn<F>();
  store.map_graphical_pointers();
  solver_df.num_particles = particle_x->read_file(argv[1]);
  store.unmap_graphical_pointers();

  U step_id = 0;
  F t = 0;
  F max_density_error = std::numeric_limits<F>::max();
  F min_density_error = std::numeric_limits<F>::max();
  bool should_close = false;

  F max_particle_speed = 99.9_F;
  F min_particle_speed = 99.9_F;
  F sum_particle_velocity_components = 99.9_F;
  F last_stationary_t = 0._F;

  std::random_device rd{};
  std::mt19937 gen{rd()};
  std::normal_distribution<F> d{0, 0.05};

  display->add_shading_program(new ShadingProgram(
      nullptr, nullptr, {}, {}, [&](ShadingProgram& program, Display& display) {
        if (should_close) {
          return;
        }

        store.map_graphical_pointers();
        for (U frame_interstep = 0; frame_interstep < 10; ++frame_interstep) {
          if (min_particle_speed > 2._F || (step_id % 10000 == 0)) {
            particle_v.set_zero();
            particle_dfsph_factor.set_zero();
            particle_kappa.set_zero();
            particle_kappa_v.set_zero();
            particle_density_adv.set_zero();
            last_stationary_t = t;
            std::cout << "last stationary t = " << last_stationary_t
                      << std::endl;
          }
          if (t > 6) {
            particle_x->write_file(argv[2], solver_df.num_particles);
            should_close = true;
          }

          solver_df.step<1, 0>();

          t += solver_df.dt;
          step_id += 1;

          max_density_error =
              Runner::max(particle_density, solver_df.num_particles) /
                  density0 -
              1;
          min_density_error =
              Runner::min(particle_density, solver_df.num_particles) /
                  density0 -
              1;
          max_particle_speed =
              sqrt(Runner::max(particle_cfl_v2, solver_df.num_particles));
          min_particle_speed =
              sqrt(Runner::min(particle_cfl_v2, solver_df.num_particles));
          sum_particle_velocity_components =
              Runner::sum<F>(particle_v.ptr_, particle_v.get_num_primitives());

          F expected_total_volume = kPi<F> * (R - particle_radius) *
                                    (R - particle_radius) * cylinder_length;
        }
        solver_df.colorize_speed(0, 2.0);
        store.unmap_graphical_pointers();
      }));

  // {{{
#include "alluvion/glsl/particle.frag"
#include "alluvion/glsl/particle.vert"
  display->add_shading_program(new ShadingProgram(
      kParticleVertexShaderStr, kParticleFragmentShaderStr,
      {"particle_radius", "screen_dimension", "M", "V", "P",
       "camera_worldspace", "material.specular", "material.shininess",
       "directional_light.direction", "directional_light.ambient",
       "directional_light.diffuse", "directional_light.specular",
       "point_lights[0].position", "point_lights[0].constant",
       "point_lights[0].linear", "point_lights[0].quadratic",
       "point_lights[0].ambient", "point_lights[0].diffuse",
       "point_lights[0].specular",
       //
       "point_lights[1].position", "point_lights[1].constant",
       "point_lights[1].linear", "point_lights[1].quadratic",
       "point_lights[1].ambient", "point_lights[1].diffuse",
       "point_lights[1].specular"

      },
      {std::make_tuple(particle_x->vbo_, 3, 0),
       std::make_tuple(particle_normalized_attr->vbo_, 1, 0)},
      [&](ShadingProgram& program, Display& display) {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glUniformMatrix4fv(
            program.get_uniform_location("P"), 1, GL_FALSE,
            glm::value_ptr(display.camera_.getProjectionMatrix()));
        glUniformMatrix4fv(program.get_uniform_location("V"), 1, GL_FALSE,
                           glm::value_ptr(display.camera_.getViewMatrix()));
        glUniform2f(program.get_uniform_location("screen_dimension"),
                    static_cast<GLfloat>(display.width_),
                    static_cast<GLfloat>(display.height_));
        glUniform1f(program.get_uniform_location("particle_radius"),
                    particle_radius);

        glm::vec3 const& camera_worldspace = display.camera_.getCenter();
        glUniform3f(program.get_uniform_location("camera_worldspace"),
                    camera_worldspace[0], camera_worldspace[1],
                    camera_worldspace[2]);
        glUniform3f(program.get_uniform_location("directional_light.direction"),
                    0.2f, 1.0f, 0.3f);
        glUniform3f(program.get_uniform_location("directional_light.ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("directional_light.diffuse"),
                    0.4f, 0.4f, 0.4f);
        glUniform3f(program.get_uniform_location("directional_light.specular"),
                    0.5f, 0.5f, 0.5f);

        glUniform3f(program.get_uniform_location("point_lights[0].position"),
                    2.0f, 2.0f, 2.0f);
        glUniform1f(program.get_uniform_location("point_lights[0].constant"),
                    1.0f);
        glUniform1f(program.get_uniform_location("point_lights[0].linear"),
                    0.09f);
        glUniform1f(program.get_uniform_location("point_lights[0].quadratic"),
                    0.032f);
        glUniform3f(program.get_uniform_location("point_lights[0].ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("point_lights[0].diffuse"),
                    0.8f, 0.8f, 0.8f);
        glUniform3f(program.get_uniform_location("point_lights[0].specular"),
                    1.0f, 1.0f, 1.0f);

        glUniform3f(program.get_uniform_location("point_lights[1].position"),
                    2.0f, 1.0f, -2.0f);
        glUniform1f(program.get_uniform_location("point_lights[1].constant"),
                    1.0f);
        glUniform1f(program.get_uniform_location("point_lights[1].linear"),
                    0.09f);
        glUniform1f(program.get_uniform_location("point_lights[1].quadratic"),
                    0.032f);
        glUniform3f(program.get_uniform_location("point_lights[1].ambient"),
                    0.05f, 0.05f, 0.05f);
        glUniform3f(program.get_uniform_location("point_lights[1].diffuse"),
                    0.8f, 0.8f, 0.8f);
        glUniform3f(program.get_uniform_location("point_lights[1].specular"),
                    1.0f, 1.0f, 1.0f);
        glUniform3f(program.get_uniform_location("material.specular"), 0.8f,
                    0.9f, 0.9f);
        glUniform1f(program.get_uniform_location("material.shininess"), 5.0f);

        glBindTexture(GL_TEXTURE_1D, colormap_tex);
        for (I i = 0; i <= 0; ++i) {
          float wrap_length = grid_res.y * kernel_radius;
          glUniformMatrix4fv(
              program.get_uniform_location("M"), 1, GL_FALSE,
              glm::value_ptr(glm::translate(glm::mat4(1),
                                            glm::vec3{wrap_length * i, 0, 0})));
          glDrawArrays(GL_POINTS, 0, solver_df.num_particles);
        }
      }));

#include "alluvion/glsl/glyph.frag"
#include "alluvion/glsl/glyph.vert"
  display->add_shading_program(new ShadingProgram(
      kGlyphVertexShaderStr, kGlyphFragmentShaderStr,
      {
          "projection",
          "text_color",
      },
      {std::make_tuple(glyph_quad, 4, 0)},
      [&](ShadingProgram& program, Display& display) {
        glm::mat4 projection =
            glm::ortho(0.0f, static_cast<GLfloat>(display.width_), 0.0f,
                       static_cast<GLfloat>(display.height_));
        glUniformMatrix4fv(program.get_uniform_location("projection"), 1,
                           GL_FALSE, glm::value_ptr(projection));
        glUniform3f(program.get_uniform_location("text_color"), 1.0f, 1.0f,
                    1.0f);

        std::stringstream time_text;
        time_text << "num_particles = " << solver_df.num_particles
                  << " t: " << std::fixed << std::setprecision(3)
                  << std::setw(6) << t << " dt: " << std::scientific
                  << std::setprecision(3) << std::setw(6) << solver_df.dt
                  << " d: (" << std::scientific << std::setprecision(3)
                  << std::setw(6) << min_density_error << "," << std::scientific
                  << std::setprecision(3) << std::setw(6) << max_density_error
                  << ") v=(" << std::setw(6) << min_particle_speed << ","
                  << max_particle_speed << ")";
        std::string text = time_text.str();

        typesetter.start(display.width_ * 0.02f, display.height_ * 0.05f, 1.0f);
        for (std::string::const_iterator c = text.begin(); c != text.end();
             c++) {
          glBindTexture(GL_TEXTURE_2D, typesetter.place_glyph(*c));
          glBindBuffer(GL_ARRAY_BUFFER, glyph_quad);
          glBufferSubData(GL_ARRAY_BUFFER, 0, sizeof(typesetter.vertices_info_),
                          typesetter.vertices_info_);
          glBindBuffer(GL_ARRAY_BUFFER, 0);
          glDrawArrays(GL_TRIANGLES, 0, 6);
        }
        glBindTexture(GL_TEXTURE_2D, 0);
      }));
  display->run();
  // }}}
}
