#include "hip/hip_runtime.h"
#include <doctest/doctest.h>

#include <vector>

#include "alluvion/constants.hpp"
#include "alluvion/dg/cubic_lagrange_discrete_grid.hpp"
#include "alluvion/dg/peirce_quadrature.hpp"
#include "alluvion/dg/sph_kernels.hpp"
#include "alluvion/float_shorthands.hpp"
#include "alluvion/runner.hpp"
#include "alluvion/store.hpp"

using namespace alluvion;
using namespace alluvion::dg;

SCENARIO("testing volume field generation") {
  GIVEN("a host implementation") {
    std::array<unsigned int, 3> resolution_array{11, 11, 11};
    F map_thickness = 0.1;
    F particle_radius = 0.25;
    F sign = 1.0;
    F r = 1.25;
    F margin = 4 * r + map_thickness;
    Vector3r<F> domain_min(-30 - margin, -20 - margin, -40 - margin);
    Vector3r<F> domain_max(30 + margin, 20 + margin, 40 + margin);
    AlignedBox3r<F> domain(domain_min, domain_max);
    CubicLagrangeDiscreteGrid<F> grid(domain, resolution_array);
    grid.addFunction(
        [sign, map_thickness, particle_radius](Vector3r<F> const &xi) {
          F signed_distance_from_ellipsoid = xi(0) * xi(0) / (25 * 25) +
                                             xi(1) * xi(1) / (15 * 15) +
                                             xi(2) * xi(2) / (35 * 35) - 1.0;
          return sign * signed_distance_from_ellipsoid;
        });
    CubicKernel<F> cubic_kernel;
    cubic_kernel.setRadius(r);
    grid.addFunction(
        [&](Vector3r<F> const &x) {
          auto dist = grid.interpolate(0u, x);
          auto integrand = [&grid, &x, &r,
                            &cubic_kernel](Vector3r<F> const &xi) -> F {
            auto dist = grid.interpolate(0u, x + xi);
            if (dist <= 0.0) return static_cast<F>(1.0);
            return cubic_kernel.W(dist) / cubic_kernel.W_zero();
          };
          return PeirceQuadrature<F>::integrate(integrand, r);
        },
        false);
    WHEN("a device implementation is constructed") {
      Runner<F> runner;
      CubicLagrangeDiscreteGrid<F> distance_grid_prerequisite(domain,
                                                              resolution_array);
      distance_grid_prerequisite.addFunction([](Vector3r<F> const &xi) {
        F signed_distance_from_ellipsoid = xi(0) * xi(0) / (25 * 25) +
                                           xi(1) * xi(1) / (15 * 15) +
                                           xi(2) * xi(2) / (35 * 35) - 1.0;
        return signed_distance_from_ellipsoid;
      });
      // copy attributes
      U num_nodes = distance_grid_prerequisite.node_data()[0].size();
      F3 domain_min = {domain.min()(0), domain.min()(1), domain.min()(2)};
      F3 domain_max = {domain.max()(0), domain.max()(1), domain.max()(2)};
      U3 resolution = make_uint3(resolution_array[0], resolution_array[1],
                                 resolution_array[2]);
      F3 cell_size = {distance_grid_prerequisite.cellSize()(0),
                      distance_grid_prerequisite.cellSize()(1),
                      distance_grid_prerequisite.cellSize()(2)};
      // allocate device memory
      Store store;
      std::unique_ptr<Variable<1, F>> distance_nodes(
          store.create<1, F>({num_nodes}));
      std::unique_ptr<Variable<1, F>> volume_nodes(
          store.create<1, F>({num_nodes}));
      distance_nodes->set_bytes(
          distance_grid_prerequisite.node_data()[0].data(),
          num_nodes * sizeof(F));

      // set constants
      store.get_cn<F>().set_particle_attr(particle_radius, 0.2, 1.0);
      store.get_cn<F>().set_kernel_radius(r);
      store.get_cn<F>().set_cubic_discretization_constants();
      store.copy_cn<F>();

      runner.launch(
          num_nodes,
          [&](U grid_size, U block_size) {
            update_volume_field<<<grid_size, block_size>>>(
                *volume_nodes, *distance_nodes, domain_min, domain_max,
                resolution, cell_size, num_nodes, sign, map_thickness);
          },
          "update_volume_field", update_volume_field<F3, F>);

      std::vector<F> device_volume_nodes_copied(num_nodes);
      volume_nodes->get_bytes(device_volume_nodes_copied.data(),
                              device_volume_nodes_copied.size() * sizeof(F));
      for (U l = 0; l < num_nodes; ++l) {
        // if (device_volume_nodes_copied[l] > 1e-5) {
        //   std::cout << l << " " << device_volume_nodes_copied[l] << " "
        //             << grid.node_data()[1][l] << std::endl;
        // }
        CHECK(device_volume_nodes_copied[l] ==
              doctest::Approx(grid.node_data()[1][l]));
      }
      store.remove(*distance_nodes);
      store.remove(*volume_nodes);
    }
  }
}
