#include <hip/hip_runtime.h>

#include <cstring>
#include <iostream>

#include "alluvion/allocator.hpp"

namespace alluvion {
Allocator::Allocator() {}
Allocator::~Allocator() {}

void Allocator::free(void** ptr) {
  if (ptr == nullptr) return;
  abort_if_error(hipFree(*ptr));
  *ptr = nullptr;
}

void Allocator::free_pinned(void** ptr) {
  if (ptr == nullptr) return;
  abort_if_error(hipHostFree(*ptr));
  *ptr = nullptr;
}

hipTextureObject_t Allocator::create_texture(
    void* ptr, size_t num_bytes, hipChannelFormatDesc const& channel_desc) {
  hipTextureObject_t tex = 0;
  if (num_bytes == 0) {
    return tex;
  }

  hipResourceDesc res_desc;
  std::memset(&res_desc, 0, sizeof(res_desc));
  res_desc.resType = hipResourceTypeLinear;
  res_desc.res.linear.devPtr = ptr;
  res_desc.res.linear.desc = channel_desc;
  if (res_desc.res.linear.desc.f == hipChannelFormatKindNone) {
    return tex;
  }
  res_desc.res.linear.sizeInBytes = num_bytes;

  hipTextureDesc tex_desc;
  std::memset(&tex_desc, 0, sizeof(tex_desc));
  tex_desc.readMode = hipReadModeElementType;

  abort_if_error(hipCreateTextureObject(&tex, &res_desc, &tex_desc, nullptr));
  return tex;
}

void Allocator::destroy_texture(hipTextureObject_t* tex) {
  if (*tex == 0) return;
  abort_if_error(hipDestroyTextureObject(*tex));
  *tex = 0;
}

void Allocator::copy(void* dst, void const* src, size_t num_bytes) {
  abort_if_error(hipMemcpy(dst, src, num_bytes, hipMemcpyDefault));
}

void Allocator::set(void* dst, size_t num_bytes, int value) {
  abort_if_error(hipMemset(dst, value, num_bytes));
}

void Allocator::get_device_properties(hipDeviceProp_t* prop, int device) {
  abort_if_error(hipGetDeviceProperties(prop, device));
}

void Allocator::abort_if_error(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << "CUDA API returns the error: " << hipGetErrorString(err)
              << std::endl;
    abort();
  }
};
}  // namespace alluvion
