#include <hip/hip_runtime.h>

#include <iostream>

#include "alluvion/allocator.hpp"

namespace alluvion {
Allocator::Allocator() {}
Allocator::~Allocator() {}

void Allocator::free(void** ptr) {
  if (ptr == nullptr) return;
  abort_if_error(hipFree(*ptr));
  *ptr = nullptr;
}

void Allocator::free_pinned(void** ptr) {
  if (ptr == nullptr) return;
  abort_if_error(hipHostFree(*ptr));
  *ptr = nullptr;
}

void Allocator::copy(void* dst, void const* src, unsigned int num_bytes) {
  abort_if_error(hipMemcpy(dst, src, num_bytes, hipMemcpyDefault));
}

void Allocator::set(void* dst, unsigned int num_bytes, int value) {
  abort_if_error(hipMemset(dst, value, num_bytes));
}

void Allocator::abort_if_error(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << "CUDA API returns the error: " << hipGetErrorString(err)
              << std::endl;
    abort();
  }
};
}  // namespace alluvion
