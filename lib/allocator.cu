#include <hip/hip_runtime.h>

#include <iostream>

#include "alluvion/allocator.hpp"

namespace alluvion {
Allocator::Allocator() {}
Allocator::~Allocator() {}

void Allocator::free(void** ptr) {
  abort_if_error(hipFree(*ptr));
  *ptr = nullptr;
}

void Allocator::copy_to_host(void* dst, void const* src,
                             unsigned int num_bytes) {
  // TODO: use hipMemcpyDefault
  abort_if_error(hipMemcpy(dst, src, num_bytes, hipMemcpyDeviceToHost));
}

void Allocator::copy_to_device(void* dst, void const* src,
                               unsigned int num_bytes) {
  abort_if_error(hipMemcpy(dst, src, num_bytes, hipMemcpyHostToDevice));
}

void Allocator::set_device(void* dst, unsigned int num_bytes, int value) {
  abort_if_error(hipMemset(dst, value, num_bytes));
}

void Allocator::abort_if_error(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << "CUDA API returns the error: " << hipGetErrorString(err)
              << std::endl;
    abort();
  }
};
}  // namespace alluvion
