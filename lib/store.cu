#include "hip/hip_runtime.h"
#include <tuple>
#include <utility>
#include <vector>

#include "alluvion/store.hpp"

namespace alluvion {
Store::Store() {}
Store::~Store() {}
void Store::update_resource_array() {
  resource_array_.resize(graphical_resource_dict_.size());
  int i = 0;
  for (auto& entry : graphical_resource_dict_) {
    resource_array_[i++] = entry.second.res_;
  }
}
Display* Store::create_display(int width, int height, const char* title) {
  display_.reset(new Display(width, height, title));
  return display_.get();
}
Display* Store::get_display() const { return display_.get(); }
bool Store::has_display() const { return static_cast<bool>(display_); }
MeshBuffer Store::create_mesh_buffer(U num_vertices, U num_faces) {
  if (!display_) {
    std::cerr << "Display not created" << std::endl;
    abort();
  }
  MeshBuffer mesh_buffer;
  mesh_buffer.num_vertices = num_vertices;
  mesh_buffer.num_indices = num_faces * 3;
  mesh_buffer.vertex =
      GraphicalAllocator::allocate_static_array_buffer<float3>(num_faces);
  mesh_buffer.index =
      GraphicalAllocator::allocate_element_array_buffer<unsigned int>(
          mesh_buffer.num_indices);
  mesh_dict_.emplace(std::piecewise_construct,
                     std::forward_as_tuple(mesh_buffer.vertex),
                     std::forward_as_tuple(mesh_buffer));
  return mesh_buffer;
}
void Store::map_graphical_pointers() {
  GraphicalAllocator::map(resource_array_);

  for (auto& entry : graphical_resource_dict_) {
    UniqueGraphicalResource& unique_resource = entry.second;
    unique_resource.var_->set_pointer(
        GraphicalAllocator::get_mapped_pointer(unique_resource.res_));
  }
}
void Store::unmap_graphical_pointers() {
  GraphicalAllocator::unmap(resource_array_);
  for (auto& entry : graphical_resource_dict_) {
    UniqueGraphicalResource& unique_resource = entry.second;
    unique_resource.var_->set_pointer(nullptr);
  }
}

}  // namespace alluvion
