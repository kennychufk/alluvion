#include "alluvion/graphical_allocator.hpp"

namespace alluvion {
GraphicalAllocator::GraphicalAllocator() {}
GraphicalAllocator::~GraphicalAllocator() {}

void GraphicalAllocator::free(GLuint* vbo, hipGraphicsResource** res) {
  if (*res == nullptr) return;
  Allocator::abort_if_error(hipGraphicsUnregisterResource(*res));
  *res = nullptr;
  if (*vbo == 0) return;
  glDeleteBuffers(1, vbo);
  *vbo = 0;
}

void GraphicalAllocator::map(std::vector<hipGraphicsResource*>& resources) {
  Allocator::abort_if_error(
      hipGraphicsMapResources(resources.size(), resources.data()));
}

void* GraphicalAllocator::get_mapped_pointer(hipGraphicsResource* res) {
  void* ptr;
  std::size_t returned_buffer_size;
  Allocator::abort_if_error(
      hipGraphicsResourceGetMappedPointer(&ptr, &returned_buffer_size, res));
  return ptr;
}

void GraphicalAllocator::unmap(std::vector<hipGraphicsResource*>& resources) {
  Allocator::abort_if_error(
      hipGraphicsUnmapResources(resources.size(), resources.data()));
}

}  // namespace alluvion
