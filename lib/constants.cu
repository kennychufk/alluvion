#include "alluvion/constants.hpp"

namespace alluvion {
__constant__ F kernel_radius;
void set_kernel_radius(F r) {
  hipMemcpyToSymbol(HIP_SYMBOL(&kernel_radius), &r, sizeof(F), 0, hipMemcpyHostToDevice);
}
}  // namespace alluvion
