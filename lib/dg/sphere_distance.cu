#include "hip/hip_runtime.h"
#include "alluvion/dg/sphere_distance.hpp"

namespace alluvion {
namespace dg {
SphereDistance::SphereDistance(F radius)
    : radius_(radius),
      Distance(F3{-radius, -radius, -radius}, F3{radius, radius, radius},
               radius) {}

F SphereDistance::signedDistance(dg::Vector3r const& x) const {
  return x.norm() - radius_;
}

}  // namespace dg
}  // namespace alluvion
